#include "hip/hip_runtime.h"
#include<iostream>
#include<sys/time.h>
#include<stdlib.h>
#include<stdio.h>
#include<hip/hip_runtime.h>

#define N 500
#define ITERATIONS 10
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8
using namespace std;

__global__ void sgemm(float *A, float *B, float *C, int n, float a, float b) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if( (i<n) && (j<n) ){
    float tmp = b*C[i*n+j];
    for(int k=0; k<n; k++){
      tmp += a*A[i*n+k]*B[k*n+j];
    }
    C[i*n+j]=tmp;
  }
}

void compare(float* res1, float* res2, int n){
  int fail=0;
  for(int i=0; i<n; i++){
    float a,b;
    if(res1[i]<0)
      a=res1[i]*(-1);
    else 
      a=res1[i];
    if(res2[i]<0)
      b=res2[i]*(-1);
    else 
      b=res2[i];
    if((a<0.01)&&(b<0.01)){
      continue;
    }
    if(i<10)
      printf("i=%d %lf %lf\n",i,a,b);
    float diff=(a-b)/(a+0.000001);
    if(diff<0)
      diff=diff*(-1);
    if(diff>0.0005)
      fail++;
  }
  printf("Number of errors: %d\n", fail);
}

double timestamp(){
  struct timeval tv;
  gettimeofday (&tv, 0);
  return tv.tv_sec + 1e-6*tv.tv_usec;
}

int main(){
  float A[N*N], B[N*N], C_cpu[N*N], C_gpu_final[N*N];
  //float A[N][N], B[N][N], C_cpu[N][N], C_gpu_final[N][N];
  float a=0.5, b=0.3;
  for(int i=0; i<N; i++){
    for(int j=0; j<N; j++){
      A[i*N+j]=(float)rand()/(float)(RAND_MAX/a);
      B[i*N+j]=(float)rand()/(float)(RAND_MAX/a);
      C_cpu[i*N+j]=0;
      C_gpu_final[i*N+j]=0;
    }
  }

  for(int j=0; j<N; j++){
    for(int i=0; i<N; i++){
      C_cpu[i*N+j]+=b*C_cpu[i*N+j];
      for(int k=0; k<N; k++){
        C_cpu[i*N+j] += a*A[i*N+k]*B[k*N+j];
      }
    }
  }

  float *A_gpu;
  float *B_gpu;
  float *C_gpu;
  hipMalloc((void **)&A_gpu, sizeof(float)*N*N);
  hipMalloc((void **)&B_gpu, sizeof(float)*N*N);
  hipMalloc((void **)&C_gpu, sizeof(float)*N*N);
  hipMemcpy(A_gpu, A, sizeof(float)*N*N, hipMemcpyHostToDevice);
  hipMemcpy(B_gpu, B, sizeof(float)*N*N, hipMemcpyHostToDevice);
  hipMemcpy(C_gpu, C_gpu_final, sizeof(float)*N*N, hipMemcpyHostToDevice);

  dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid((size_t)ceil( ((float)N) / ((float)block.x) ), (size_t)ceil( ((float)N) / ((float)block.y)) );

  sgemm<<<grid,block>>>(A_gpu, B_gpu, C_gpu, N, a, b);
  hipDeviceSynchronize();
  hipMemcpy(C_gpu_final, C_gpu, sizeof(float)*N*N, hipMemcpyDeviceToHost);
  compare(C_cpu, C_gpu_final, N*N);

  double time1=timestamp();
  for(int numOfTimes=0; numOfTimes<ITERATIONS; numOfTimes++){

    sgemm<<<grid,block>>>(A_gpu, B_gpu, C_gpu, N, a, b);

  }
  hipDeviceSynchronize();
  double time2=timestamp();

  double time = (time2-time1)/ITERATIONS;
  double flops = 2*N*N*N;
  double gflopsPerSecond = flops/(1000000000)/time;
  double GB = (double)(N)*N*4/1000000000;
  double GBpS = (double)(N)*N*4/1000000000/time;
  printf("GFLOPS/s=%lf\n",gflopsPerSecond );
  printf("GB/s=%lf\n",GBpS);
  printf("GFLOPS=%lf\n",flops/(1000000000));
  printf("GB=%lf\n",GB);
  printf("time(s)=%lf\n",time);

  hipFree(A_gpu);
  hipFree(B_gpu);
  hipFree(C_gpu);
  return 0;
}
